
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <vector>
#include <iostream>

__global__ void VecAdd(float* A, float* B, float* C)
{
    int x = threadIdx.x;
    int y = threadIdx.y;

    C[x+y*32] = x*y;
}

int main()
{
    int N = 32;
    float *a, *b, *c;
    float *a_gpu, *b_gpu, *c_gpu;

    a = (float *) malloc(sizeof(float) * N);
    b = (float *) malloc(sizeof(float) * N);
    c = (float *) malloc(sizeof(float) * N * N);

    for (int i = 0; i < N; ++i) 
    {
        a[i] = i; 
        b[i] = i; 
    }


    hipMalloc((void **) &a_gpu, sizeof(float) * N);
    hipMalloc((void **) &b_gpu, sizeof(float) * N);
    hipMalloc((void **) &c_gpu, sizeof(float) * N * N);

    hipMemcpy(a_gpu, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b, sizeof(float) * N, hipMemcpyHostToDevice);

    dim3 gridDim;
    dim3 blockDim(32,32);
    VecAdd<<<gridDim, blockDim>>>(a_gpu, b_gpu, c_gpu);

    hipMemcpy(c, c_gpu, sizeof(float) * N * N, hipMemcpyDeviceToHost);

    for (int y = 0; y < N; ++y) 
    {
        for (int x = 0; x < N; ++x) 
        {
            std::cout << c[x+y*32] << "|";
        }
        std::cout << std::endl;
    }

    free(a);
    free(b);
    free(c);

    hipFree(a_gpu);
    hipFree(b_gpu);
    hipFree(c_gpu);
}

